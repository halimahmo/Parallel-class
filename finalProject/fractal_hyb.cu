/*
Fractal code for CS 4380 / CS 5351

Copyright (c) 2018, Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>

static const int ThreadsPerBlock = 512;

static const double Delta = 0.004;
static const double xMid =  0.2389;
static const double yMid =  0.55267;

static __global__ void FractalKernel(const int start_frame, const int gpu_frames, const int width, unsigned char* pic_d)
{
  // todo: use the GPU to compute the frames (base the code on the previous project)
}

unsigned char* GPU_Init(const int gpu_frames, const int width)
{
  unsigned char* pic_d;
  if (hipSuccess != hipMalloc((void **)&pic_d, gpu_frames * width * width * sizeof(unsigned char))) {fprintf(stderr, "could not allocate memory\n"); exit(-1);}
  return pic_d;
}

void GPU_Exec(const int start_frame, const int gpu_frames, const int width, unsigned char* pic_d)
{
  // todo: launch the kernel with ThreadsPerBlock and the appropriate number of blocks
}

void GPU_Fini(const int gpu_frames, const int width, unsigned char* pic, unsigned char* pic_d)
{
  // todo: copy the result from the device to the host and free the device memory
}
