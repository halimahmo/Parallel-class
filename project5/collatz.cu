#include "hip/hip_runtime.h"
/*
Collatz code for CS 4380 / CS 5351

Copyright (c) 2018, Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <sys/time.h>

static const int ThreadsPerBlock = 512;

static __global__ void collatzKernel(int* maxlen, const long range)
{
  // compute sequence lengths
  const long idx = threadIdx.x + blockIdx.x * (long)blockDim.x;
  if(idx < range){
    long val = idx;
    int len = 1;
    while (val != 1) {
      len++;
      if ((val % 2) == 0) {
        val = val / 2;  // even
      } else {
        val = 3 * val + 1;  // odd
      }
    }
    maxlen = atomicMax(maxlen, len);
    }
}

static void CheckCuda()
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
    exit(-1);
  }
}

int main(int argc, char *argv[])
{
  printf("Collatz v1.0\n");

  // check command line
  if (argc != 2) {fprintf(stderr, "usage: %s range\n", argv[0]); exit(-1);}
  const long range = atol(argv[1]);
  if (range < 1) {fprintf(stderr, "error: range must be at least 1\n"); exit(-1);}
  printf("range: 1, ..., %ld\n", range);

  //allocating space for device copy of maxlen 
  int* d_maxlen;
  const int size = range * sizeof(int);
  hipMalloc((void **)&d_maxlen, size);

  //intializing the cpu maxlen 
  int maxlen = 0;

  //copying maxlen value to device
  if (hipSuccess != hipMemcpy(d_maxlen, maxlen, size, hipMemcpyHostToDevice)) {fprintf(stderr, "copying to device failed\n"); exit(-1);}

  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // launch GPU kernel
  collatzKernel<<<(range + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(d_maxlen, range);
  hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
  printf("compute time: %.3f s\n", runtime);
  CheckCuda();

  // copy result back to host
  if (hipSuccess != hipMemcpy(&maxlen, d_maxlen, size, hipMemcpyDeviceToHost)) {fprintf(stderr, "copying from device failed\n"); exit(-1);}

  // print result
  printf("longest sequence: %d elements\n", maxlen);

  hipFree(d_maxlen);
  return 0;
}

